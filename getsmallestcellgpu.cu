#include "hip/hip_runtime.h"
#include "getsmallestcellgpu.cuh"
#include <iostream>

__global__ void find_mininum(PDistCell *secVec, vector<PDistCellMin> *min)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; // Y - ID
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; // X - ID

	float dist = secVec[i][j].dist;

	if (dist < min.dist) {
		mins.clear();
		mins.push_back(secVec[i][j]);
	}
}